/* ==================================================================
	Programmer: Yicheng Tu (ytu@cse.usf.edu)
	The basic SDH algorithm implementation for 3D data
	To compile: nvcc SDH.c -o SDH in the rc machines
   ==================================================================
*/

/*==================================================================
Anthony Long 
U53579009
Project 1, Fall 2019
run on the c4 machine 7

to run:
$ module load apps/cuda/7.5
$ nvcc proj1-anthonylong.cu -o proj1
$ /proj1 10000 500.0
  ==================================================================
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define BOX_SIZE	23000 /* size of the data box on one dimension            */

/* descriptors for single atom in the tree */
typedef struct atomdesc 
{
	double x_pos;
	double y_pos;
	double z_pos;
} atom;

typedef struct hist_entry
{
	//float min;
	//float max;
	unsigned long long d_cnt;   /* need a long long type as the count might be huge */
} bucket;



bucket *GPUhistogram;	/* list of all buckets in the GPUhistogram   */
bucket *CPUhistogram;	/* list of all buckets in the CPUhistogram   */	
long long PDH_acnt;		/* total number of data points            */
int num_buckets;		/* total number of buckets in the GPUhistogram */
double PDH_res;			/* value of w                             */
atom *atom_list;		/* list of all data points                */


double p2p_distanceCPU(int ind1, int ind2) {
	
	double x1 = atom_list[ind1].x_pos;
	double x2 = atom_list[ind2].x_pos;
	double y1 = atom_list[ind1].y_pos;
	double y2 = atom_list[ind2].y_pos;
	double z1 = atom_list[ind1].z_pos;
	double z2 = atom_list[ind2].z_pos;
		
	return sqrt((x1 - x2)*(x1-x2) + (y1 - y2)*(y1 - y2) + (z1 - z2)*(z1 - z2));
}

//__devicce__ means function is executed on GPU, rather than the CPU
__device__ double p2p_distance(atom *p, int ind1, int ind2) 
{
	double x1 = p[ind1].x_pos;
	double x2 = p[ind2].x_pos;
	double y1 = p[ind1].y_pos;
	double y2 = p[ind2].y_pos;
	double z1 = p[ind1].z_pos;
	double z2 = p[ind2].z_pos;
		
	return sqrt((x1 - x2)*(x1 - x2) + (y1 - y2)*(y1 - y2) + (z1 - z2)*(z1 - z2));
}

/* 
	brute-force SDH solution in a single CPU thread 
*/
int PDH_baselineCPU() {
	int i, j, h_pos;
	double dist;
	
	for(i = 0; i < PDH_acnt; i++) {
		for(j = i+1; j < PDH_acnt; j++) {
			//printf("i: %d, j: %d \n", i, j);
			dist = p2p_distanceCPU(i,j);
			h_pos = (int) (dist / PDH_res);
			//printf("dis: %f, W: %f, pos: %d \n", dist, PDH_res, h_pos);

			CPUhistogram[h_pos].d_cnt++;
		} 
	}
	return 0;
}

__global__ void PDH_baseline(bucket *hist, atom *atomList, double width, int size) 
{
	int i, pos;
	double dis;
	
	//i is computed by taking the correct block times the dimention of the block + the thread offset
	i = (blockIdx.x * blockDim.x) + threadIdx.x;

	/*iterates through each n-1 point pairs, using p2p distance 
	to find distance, divide it by the w, and use atomic add to incriment the correct bucket in the histogram*/
	for (int j = i+1; j < size; ++j) 
	{
		dis = p2p_distance(atomList, i, j);
		pos = (int) (dis / width);
		atomicAdd( &hist[pos].d_cnt, 1);
		__syncthreads();
	}
}

void output_histogram(bucket *hist){
	int i; 
	long long total_cnt = 0;
	for(i=0; i< num_buckets; i++) {
		if(i%5 == 0) /* we print 5 buckets in a row */
			printf("\n%02d: ", i);
		printf("%15lld ", hist[i].d_cnt);
		total_cnt += hist[i].d_cnt;
	  	/* we also want to make sure the total distance count is correct */
		if(i == num_buckets - 1)	
			printf("\n T:%lld \n", total_cnt);
		else printf("| ");
	}
	printf("\n");
}

void output_differences(bucket *hist1, bucket *hist2){
	int i; 
	long long total_cnt = 0;
	for(i=0; i< num_buckets; i++) {
		if(i%5 == 0) /* we print 5 buckets in a row */
			printf("\n%02d: ", i);
		int diff = 	hist1[i].d_cnt - hist2[i].d_cnt;
		printf("%15lld ", diff);
		total_cnt += diff;
	  	/* we also want to make sure the total distance count is correct */
		if(i == num_buckets - 1)	
			printf("\n Total differences:%lld \n", total_cnt);
		else printf("| ");
	}
	printf("\n");
}


int main(int argc, char const *argv[])
{
	PDH_acnt = atoi(argv[1]);	// Number of atoms
	PDH_res = atof(argv[2]);	// Input Distance: W

	num_buckets = (int)(BOX_SIZE * 1.732 / PDH_res) + 1;

	//sizeof for malloc allocation
	size_t histSize = sizeof(bucket)*num_buckets;
	size_t atomSize = sizeof(atom)*PDH_acnt;

	//host histograms and atomlist
	GPUhistogram = (bucket *)malloc(histSize);
	CPUhistogram = (bucket *)malloc(histSize);
	atom_list = (atom *)malloc(atomSize);

	srand(1);
	/* generate data following a uniform distribution */
	for(int i = 0;  i < PDH_acnt; i++) 
	{
		atom_list[i].x_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
		atom_list[i].y_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
		atom_list[i].z_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
	}

	//device histogram and atom list
	bucket *d_histogram = NULL;
	atom *d_atom_list = NULL;

	//cuda malloc
	hipMalloc((void**) &d_histogram, histSize);
	hipMalloc((void**) &d_atom_list, atomSize);

	//cuda mem copy to device
	hipMemcpy(d_histogram, GPUhistogram, histSize, hipMemcpyHostToDevice);
	hipMemcpy(d_atom_list, atom_list, atomSize, hipMemcpyHostToDevice);

	//cpu baseline and gpu baseline
	PDH_baselineCPU();
	PDH_baseline <<<ceil(PDH_acnt/32.0), 32>>> (d_histogram, d_atom_list, PDH_res, PDH_acnt);

	//output of cpu, gpu, and differences
	hipMemcpy(GPUhistogram, d_histogram, histSize, hipMemcpyDeviceToHost);
	printf("CPU:");
	output_histogram(CPUhistogram);
	printf("GPU:");
	output_histogram(GPUhistogram);
	printf("Differences:");
	output_differences(CPUhistogram, GPUhistogram );

	//cuda mem free
	hipFree(d_histogram);
	hipFree(d_atom_list);

	//free allocated
	free(GPUhistogram);
	free(CPUhistogram);
	free(atom_list);

	hipDeviceReset();

	return 0;
}









